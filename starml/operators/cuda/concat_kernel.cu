#include "hip/hip_runtime.h"
#include "starml/operators/concat.h"
#include "starml/basic/common_cuda.h"

namespace starml {
namespace {
template <typename T>
__global__ void concat_kernel(T* data_1, T* data_2, T* res_data, int size, int w1, int w2) {
  int pos = blockDim.x * blockIdx.x + threadIdx.x;
  if (pos < size) {
      int n = pos / (w1 + w2);
      int m = pos % (w1 + w2);
      res_data[pos] = m >= w1 ? data_2[n * w2 + m - w1] : data_1[n * w1 + m];
  }
}

void concat_impl(const Matrix& matrix1, const Matrix& matrix2, Matrix& result, int axis) {
// void concat_impl(const Matrix& matrix1, const Matrix& matrix2, Matrix& result) {
  // int axis = 1; // temp
  auto m1_rows_num =  matrix1.dim(0);
  auto m1_cols_num =  matrix1.dim(1);
  auto m2_rows_num =  matrix2.dim(0);
  auto m2_cols_num =  matrix2.dim(1);
  auto w1 = axis == 0 ? m1_rows_num : m1_cols_num;
  auto w2 = axis == 0 ? m2_rows_num : m2_cols_num;
  auto data_type = result.data_type().type();
  auto size = result.size();
  STARML_DISPATCH_TYPES(data_type, "CONCAT", [&]() {
    scalar_t *data_1 = matrix1.data<scalar_t>();
    scalar_t *data_2 = matrix2.data<scalar_t>();
    scalar_t *res_data = result.data<scalar_t>();
    dim3 dimGrid(ceil(size / 256.0), 1, 1);
    dim3 dimBlock(256, 1, 1);
    concat_kernel<scalar_t><<<dimGrid, dimBlock>>>(data_1, data_2, res_data, size, w1, w2);
  });
}
}  // namespace
STARML_REGISTER_KERNEL(concat_dispatcher, kCUDA, &concat_impl);

}  // namespace starml
