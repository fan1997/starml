#include "starml/operators/transpose.h"
#include "starml/basic/common_cuda.h"
#include <iostream>
namespace starml {
namespace {

void trans_impl(const Matrix& matrix1, Matrix& result) {
  auto data_type = matrix1.data_type().type();
  int rows_num = matrix1.dim(0);
  int cols_num = matrix1.dim(1);
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  switch (data_type) {
      case kInt:

      case kFloat:{
        using scalar_t = float;
        const scalar_t *data1 = matrix1.data<scalar_t>();
        scalar_t *res_data = result.data<scalar_t>();
        scalar_t alpha = 1.;
        scalar_t beta  = 0.;
        hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, rows_num, cols_num, &alpha, data1,
                    cols_num, &beta, data1, cols_num, res_data, rows_num);
        break;
      }
      case kDouble:{
        using scalar_t = double;
        const scalar_t *data1 = matrix1.data<scalar_t>();
        scalar_t *res_data = result.data<scalar_t>();
        scalar_t alpha = 1.;
        scalar_t beta  = 0.;
        hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, rows_num, cols_num, &alpha, data1,
                    cols_num, &beta, data1, cols_num, res_data, rows_num);
        break;
      }
      default:
         std::cout << "This function doesn't handle types other than "
                     "float, double";
  }

}
}  // namespace
STARML_REGISTER_KERNEL(transpose_dispatcher, kCUDA, &trans_impl);
}  // namespace starml
