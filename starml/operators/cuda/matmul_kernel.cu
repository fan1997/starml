#include "starml/operators/matmul.h"
#include "starml/basic/common_cuda.h"

namespace starml {
namespace {

void matmul_impl(const Matrix& matrix1, const Matrix& matrix2,  Matrix& result) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  auto data_type = matrix1.data_type().type();
  const int n_rows_mat1 = matrix1.rows_num();
  const int n_cols_mat1 = matrix1.cols_num();
  const int n_rows_mat2 = matrix2.rows_num();
  const int n_cols_mat2 = matrix2.cols_num();
  const int m = n_cols_mat2;
  const int k = n_rows_mat2;
  const int n = n_rows_mat1;
  const int lda = n_cols_mat2;
  const int ldb = n_rows_mat2;
  const int ldc = n_cols_mat2;
  switch (data_type) {
      case kFloat:{
        using scalar_t = float;
        const scalar_t *data1 = matrix2.data<scalar_t>();
        const scalar_t *data2 = matrix1.data<scalar_t>();
        scalar_t *res_data = result.data<scalar_t>();
        scalar_t alpha = 1.;
        scalar_t beta  = 0.;
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, data1, lda, data2, ldb,
                    &beta, res_data, ldc);
        break;
      }
      case kDouble:{
        using scalar_t = double;
        const scalar_t *data1 = matrix2.data<scalar_t>();
        const scalar_t *data2 = matrix1.data<scalar_t>();
        scalar_t *res_data = result.data<scalar_t>();
        scalar_t alpha = 1.;
        scalar_t beta  = 0.;
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, data1, lda, data2, ldb,
            &beta, res_data, ldc);
        break;
      }
      default:
         std::cout << "This function doesn't handle types other than "
                     "float, double";
  }

}
}  // namespace
STARML_REGISTER_KERNEL(matmul_dispatcher, kCUDA, &matmul_impl);
}  // namespace starml
