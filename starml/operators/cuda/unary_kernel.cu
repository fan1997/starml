#include "hip/hip_runtime.h"
#include "starml/operators/unary_ops.h"
#include "starml/basic/common_cuda.h"
#include "starml/basic/context_cuda.h"

namespace starml {
namespace {
template <typename TScalarType, typename TOp>
__global__ void unary_kernel(const TScalarType* data, int start, int end,
                             TOp op, TScalarType* result_data) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i + start < end) {
    *(result_data + i + start) = op(*(data + i + start));
  }
}

template <typename TScalarType, typename TOp>
void eval_unary(const TScalarType* data, TScalarType* result_data, int start,
                int end, HIPContext* cuda_ctx, TOp op) {
  dim3 dimGrid(ceil((end - start) / 256.0), 1, 1);
  dim3 dimBlock(256, 1, 1);
  unary_kernel<TScalarType><<<dimGrid, dimBlock, 0, cuda_ctx->stream()>>>(
      data, start, end, op, result_data);
}

void exp_impl(const Matrix& matrix, Matrix& result) {
  auto data_type = matrix.data_type().type();
  auto cuda_ctx = get_cuda_context(matrix.device());
  STARML_DISPATCH_FLOATING_TYPES(data_type, "CUDA_EXP", [&]() {
    auto data = matrix.data<scalar_t>();
    auto result_data = result.mutable_data<scalar_t>();
    cuda_ctx->prefetch_async(const_cast<scalar_t*>(data),
                             sizeof(scalar_t) * matrix.size(),
                             cuda_ctx->stream());
    cuda_ctx->prefetch_async(result_data, sizeof(scalar_t) * result.size(),
                             cuda_ctx->stream());
    eval_unary<scalar_t>(
        data, result_data, 0, result.size(), cuda_ctx,
        [=] __device__(scalar_t a) -> scalar_t { return ::exp(a); });
    cuda_ctx->synchronize();
  });
}

}  // namespace
STARML_REGISTER_KERNEL(exp_dispatcher, kCUDA, &exp_impl);
}  // namespace starml